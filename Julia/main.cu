#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <iomanip>

#include <cudaErrors.h>
#include <thrust/complex.h>
#include <ImageHelper.h>

__global__ void Julia(unsigned int nx,unsigned int ny,double centerx,double centery,double scale_x,double scale_y,PixelInfo *img){

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	const int loops = 200;
	const int strength = 30;


	if(i >= nx || j >= ny){
		return;
	}

	double lowx = centerx - scale_x/2;
	double lowy = centery - scale_y/2;

	thrust::complex<double> c = thrust::complex<double>(-1,0);
	thrust::complex<double> z = thrust::complex<double>(lowx+i/double(nx)*scale_x , lowy+j/double(ny)*scale_y);

	double mag;
	for(int l = 0; l < loops ; l++) {

		z = z*z + c;
		mag = norm(z) ;
		if( mag > 4 ) {
			img[j*nx+i].r = (l*strength/2)%256;
			img[j*nx+i].g = (l*strength)%256;
			img[j*nx+i].b = (l*strength/3)%256;
			return;
		}
	}

	img[j*nx+i].r = 256;
	img[j*nx+i].g = 256;
	img[j*nx+i].b = 256;
}


int main() {

	unsigned int nx = 1200;
	unsigned int ny = 600;
	unsigned int tx = 8;
	unsigned int ty = 4;	

	dim3 blocks(nx/tx+1,ny/ty+1);
	dim3 threads(tx,ty);

	// Alloc Img //
	PixelInfo *img;
	checkCudaErrors(hipMallocManaged((void **)&img,nx*ny*sizeof(PixelInfo)));

	double count = 10000;

	double centerx = 0.618;
	double centery = 0.0001;

	double img_count;
	double reduce;
	for(img_count=0,reduce = 1; reduce < count+1; reduce*=1.2,img_count++ ){
		// Call //
		Julia<<<blocks,threads>>>(nx,ny,centerx,centery,4.0f/reduce,2.0f/reduce,img);
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());

		// Save Img //
		std::stringstream ss;
		ss << "./save_folder/Img-" << std::setfill('0') << std::setw(5) << img_count << ".jpg";
		
		saveImage(img,nx,ny,ss.str().c_str());
	}

	// Clean Up //
	checkCudaErrors(hipFree(img));
	return 0;
}